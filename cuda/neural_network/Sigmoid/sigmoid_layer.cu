#include "hip/hip_runtime.h"
#include<math.h>
#include<iostream>
#include<cstdlib>

#define BLOCKSIZE 256


// __global__
// void dot(float* X, float* Y, float* c){
//   // int index = blockIdx.x * blockDim.x + threadIdx.x;
//   // __shared__ float cache[BLOCKSIZE];
//   //
//   // cache[threadIdx.x] = X[index] * Y[index];
//   //
//   // __syncthreads();
//   //
//   // if(threadIdx.x == 0){
//   //   int sum = 0;
//   //   for(int i = 0; i < BLOCKSIZE; i++){
//   //     sum += cache[i];
//   //   }
//   //   atomicAdd(c, sum);
//   // }
// }
//
__global__
void dot(float* X, float* Y, float* c, int n){
  __shared__ float cache[BLOCKSIZE];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  while(i < n){
    cache[threadIdx.x] = X[i] * Y[i];
    i+= gridDim.x * blockDim.x;
  }
  __syncthreads();

  i = BLOCKSIZE / 2;

  while(i > 2){
    if(threadIdx.x < i)
    cache[threadIdx.x] += cache[threadIdx.x +i];
    __syncthreads();
    i /= 2;
  }
  if (threadIdx.x ==0) atomicAdd(c, cache[0]);

}
__device__
float sigmoid(float x){
  return 1.0f / (1 + exp(-x));
}
__global__
void sigPass(int n, float *X, float *S){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = idx; i<n; i += stride){
    S[i] = sigmoid(X[i]);
  }
}
__global sigBackProp(int n, float *X, float *errors, float *out_slopes){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < n; i+= stride){
    out_slopes[i] = errors[i] * sigmoid(X[i]) * (1 - sigmoid(X[i]));
  }
}

int main(void){
  int xdim = 1<<16;
  int ydim = 1;
  int n = xdim * ydim;
  float *X, *Y, *S, *c;

  hipMallocManaged(&X, n*sizeof(float));
  hipMallocManaged(&S, sizeof(float));
  hipMallocManaged(&Y, n*sizeof(float));
  hipMallocManaged(&c, sizeof(float));

  for (int i = 0; i < n; i++){
    X[i] = (rand() %100)/10000.f;
    Y[i] = (rand() %100)/10000.f;
  }



  int numBlocks = (n + BLOCKSIZE - 1) / BLOCKSIZE;
  dot<<<numBlocks, BLOCKSIZE>>>(X,Y,c,n);
  hipDeviceSynchronize();
  printf(" dot output: %f", *c);
  hipFree(X);
  hipFree(Y);
  sigPass<<<numBlocks, BLOCKSIZE>>>(1,c,S);
  hipDeviceSynchronize();


  printf("final output: %f\n", *S);
  hipFree(c);
  hipFree(S);



}
