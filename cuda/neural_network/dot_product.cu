#include "hip/hip_runtime.h"
#include<math.h>
#include<iostream>

__global__
void dot(float* X, float* Y, float* c){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x;
  __shared__ float cache[blockDim.x];

  cache[threadIdx.x] = X[index] * Y[index];

  __syncthreads();

  if(threadIdx.x == 0){
    int sum = 0;
    for(int i = 0; i < blockDim.x; i++){
      sum += temp[i];
    }
    atomicAdd(c, sum);
  }
}
