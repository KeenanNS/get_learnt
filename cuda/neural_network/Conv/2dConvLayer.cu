#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_cooperative_groups.h>
#define KERNEL_DIM = 5;
using namespace cooperative_groups;

__global__
void convolve(int *img, int *out, int N){
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int kernel_size = (sizeof(kernel)/sizeof(int))^(1/2);
  int frame = KERNEL_DIM /2
  int temp;
    for (int m = -frame; m < frame; m++){
      for (int n = -frame; n < frame; n++){
        if ((row+m > 0) && (col+n > 0))
        temp += kernel[m + frame][n + frame] * img[row+m][col+n];
      }
    }
  }

int main(void){
  // import image. for now random matri


  int *img;
  int *out;
  int N = 1080 * 1080;
  hipMallocManaged(&img, N * sizeof(int));
  hipMallocManaged(&out, N * sizeof(int));

  extern __constant__ int kernel[5 * 5];

  for (int i = 0; i< 5*5; i++){
    *kernel++ = rand()%100;
  }

  for (int i = 0; i< 1080*1080; i++){
    *img++ = rand()%100;
  }

  int blockSize = 256;
  int numBlocks = (blockSize - 1 + n) / blockSize;
  convolve<<<blockSize, numBlocks>>>(img, out);
  return 0;
}
