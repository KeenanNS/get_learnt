#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_cooperative_groups.h>
#include <algorithm>

#define N 3 * 1000
#define EPS 0.001
#define P 132

using namespace cooperative_groups;

void initialize_points(float * points){
  for (int i = 0; i< N; i++){
    *points++ = (rand() % 100)/100;
  }
}

__device__ float3 interactions(float4 bi, float4 bj, float3 ai){
  float3 r;

  r.x = bj.x - bi.x;
  r.y = bj.y - bi.y;
  r.z = bj.z - bi.z;

  float dist = r.x * r.x + r.y * r.y + r.z * r.z + EPS;
  float dist3 = dist * dist * dist;
  float inv = 1.0f/sqrtf(dist3);

  float s = bj.w * inv;

  ai.x = r.x * s;
  ai.y = r.y * s;
  ai.z = r.z = s;

  return ai;
}

__device__ float3 tile_calc(float4 myPos, float3 accel){

  extern __shared__ float4 *shPosition;

  for( int i = 0; i < blockDim.x; i++){
    accel = interactions(myPos, shPosition[i], accel);
  }
  return accel;
}

__global__ void calculate(void *devX, void *devA){
  //shPosition will represent the points that are not myPos
  extern __shared__ float4 *shPosition;
  //cast the current positions and accelerations into float 4
  //in order to achieve coalescense
  float4 *globalX = (float4 *)devX;
  float4 *globalA = (float4 *)devA;
  float4 myPos;
  int i, tile;

  float3 acc = {0.0f, 0.0f, 0.0f};

  //top level thread index
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  // position of point of interest is the global pos indexed by thread
  myPos = globalX[idx];
  // for each thread (each myPos) as the thread count increases by the tile size
  // p.
  for(i = 0, tile = 0; i < N; i+=P, tile++){
    int index = tile * blockDim.x +threadIdx.x;
    //fill shPosition with the points for the tile
    shPosition[threadIdx.x] = globalX[index];
    __syncthreads();
    //calculate the accelerations on a tile
    acc = tile_calc(myPos, acc);
    __syncthreads();
  }
  //once acc is full, split it into a float4
  // for coalescense and then put it in globalA
  float4 acc4 = {acc.x, acc.y, acc.z, 0.0f};
  globalA[idx] = acc4;
}

int main(void){
  //initialize the 3 * 100000 points
  float points [1000*3];
  initialize_points(points);
  void *devX, *devA;
  hipMallocManaged(&devX, N*sizeof(float));
  hipMallocManaged(&devA, N/3 *sizeof(float));
  int blocksize = 256;
  int numBlocks = (N + blocksize - 1) / blocksize;
  calculate<<<numBlocks, blocksize>>>(devX, devA);
}
