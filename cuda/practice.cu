
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int count = 0;
  for (int i = index; i < n; i += stride){
    printf("count : %d\n",count ++);
    y[i] = x[i] + y[i];
  }

}

int main(void)
{
  int N = 1<<12;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  int blocksize = 256;
  int numBlocks = (N + blocksize - 1) / blocksize;
  for(int i = 0; i<100; i++)
  add<<<numBlocks, blocksize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)

  hipFree(x);
  hipFree(y);

  return 0;
}
