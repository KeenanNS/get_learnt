#include "hip/hip_runtime.h"
#include<iostream>
#include<math.h>
#include<hip/hip_cooperative_groups.h>

using namespace cooperative_groups;
int numBlocks = 40;
int blockSize = 256;

__device__ int reduce_sum(thread_group g, int *temp, int val){
  for (int i = g.size()/2; i>0; i /=2){
    // make a val variable for each thread and take the value from the shared array
    temp[g.thread_rank()] = val;
    //sync them to prevent the race condition
    g.sync();
    if(g.thread_rank() < i) {val += temp[g.thread_rank() +i];}
    g.sync();
  }
  return val;
}
__device__ int block_sum(float *input, int n){
  int sum = 0;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < n/4; i += stride){
    int4 in = ((int4*) input)[i];
    sum += in.x + in.y + in.z + in.w;
  }
  return sum;
}

__global__ void sum_kernel(float *sum, float *input, int n){
  int my_sum = block_sum(input, n);
  //shared memory, need to synchronize threads before reading or writing
  //
  extern __shared__ int temp[];
  auto g = this_thread_block();
  int block_sum = reduce_sum(g, temp, my_sum);
  if (g.thread_rank() ==0){ atomicAdd(sum, block_sum);}
}

int main(void){

  int n = 1 << 10;
  int blockSize = 256;
  int numBlocks = (n + blockSize - 1) / blockSize;
  float *input, *sum;
  hipMallocManaged(&input, n * sizeof(float));
  hipMallocManaged(&sum, sizeof(float));

  for (int i = 0; i < n; i ++){
    input[i] = (rand() %100) / 100.0;
  }

  sum_kernel <<<numBlocks, blockSize>>>(sum, input, n);
  printf("the final sum: %f", *sum);
}
